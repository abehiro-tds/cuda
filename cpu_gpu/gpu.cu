
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N 2048
#define BS 32

long start_time,end_time;
int ha[N*N],hb[N*N],hc[N*N];

__global__ void transpose(int *a,int *b, int *c){
  int i;
  int idx=blockDim.x*blockIdx.x+threadIdx.x;
  int idy=blockDim.y*blockIdx.y+threadIdx.y;
  int id =idy*N+idx;

  c[id]=0;
  for(i=0;i<N;i++)
    c[id]+=a[idy*N+i] * b[(i*N) + idx];
}

void init_array(int *data){
  for(int i=0;i<N*N;i++){
    data[i]=rand();
  }
}
  

int main(int argc, char *argv[]){
  int *da,*db,*dc;

  hipMalloc(&da,N*N*sizeof(int));
  hipMalloc(&db,N*N*sizeof(int));
  hipMalloc(&dc,N*N*sizeof(int));
  init_array((int*)ha);
  init_array((int*)hb);

  start_time=clock();
  hipMemcpy(da,(int*)ha, N*N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(db,(int*)hb, N*N*sizeof(int), hipMemcpyHostToDevice);
  dim3 dimGrid(N/BS, N/BS);
  dim3 dimBlock(BS, BS);
  transpose<<<dimGrid, dimBlock>>>(da,db,dc);
  hipMemcpy((int*)hc, dc, N*N*sizeof(int), hipMemcpyDeviceToHost);
  end_time=clock();

  printf("time:%f\n",(double)(end_time-start_time)/CLOCKS_PER_SEC);

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  return 0;
}

