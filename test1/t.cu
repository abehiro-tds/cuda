
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define SIZE 256

__global__ void arrayadd(float *fOut, float *fInA, float *fInB){
  int id = threadIdx.x+blockIdx.x*blockDim.x;
  fOut[id]=fInA[id]+fInB[id];
}

int main(int argc, char** argv){
  int i;
  printf("GPU:\n");
  srand(0);
  hipSetDevice(0);
  float *h_InA, *h_InB, *h_Out;
  h_InA=(float*)malloc(sizeof(float)*SIZE);
  h_InB=(float*)malloc(sizeof(float)*SIZE);
  h_Out=(float*)malloc(sizeof(float)*SIZE);
  for(i=0; i<SIZE; i++)   h_InA[i]=(float)(rand()%10)/10.0f;
  for(i=0; i<SIZE; i++)   h_InB[i]=(float)(rand()%10)/10.0f;
  printf("InA: "); for(i=0; i<SIZE; i++)printf(" %.2f",h_InA[i]); printf("\n");
  printf("InB: "); for(i=0; i<SIZE; i++)printf(" %.2f",h_InB[i]); printf("\n");

  float *d_InA, *d_InB, *d_Out;
  hipMalloc((void**)&d_InA, sizeof(float)*SIZE);
  hipMalloc((void**)&d_InB, sizeof(float)*SIZE);
  hipMalloc((void**)&d_Out, sizeof(float)*SIZE);
  hipMemcpy(d_InA, h_InA, sizeof(float)*SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_InB, h_InB, sizeof(float)*SIZE, hipMemcpyHostToDevice);

  arrayadd<<< 16, 16 >>>(d_Out, d_InA, d_InB);

  hipMemcpy(h_Out, d_Out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
  printf("Out: "); for(i=0; i<SIZE; i++)printf(" %.2f", h_Out[i]); printf("\n");

  free(h_InA); free(h_InB); free(h_Out);

  hipFree(d_InA); hipFree(d_InB); hipFree(d_Out);
  return 0;
}
